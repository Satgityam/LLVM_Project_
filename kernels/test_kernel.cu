
#include <hip/hip_runtime.h>
__global__ void raceKernel(int *data) {
  int idx = threadIdx.x;
  data[idx] = idx;  // Write
  int val = data[idx];  // Read
}

int main() {
  int *d_data;
  hipMalloc((void**)&d_data, sizeof(int) * 256);
  raceKernel<<<1, 256>>>(d_data);
  hipFree(d_data);
  return 0;
}
